#include "hip/hip_runtime.h"
//program description:
//brwonian paritcles dynamics in a box use GPU
/*
kernel code
first get CellList
then get around particle id per particle
third get hybrid list use bit calculate
firth get force
last update position
*/
//use a flag to mask whether cell of idth particle at the edge of box, use -1 0 1, if at right FlagX=1;
//In the end of every function,there is a __syncthreads to pervent error
//����csv��Ϊ�������ݵ��ļ���ʽ

#include <hipfft/hipfftXt.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <stdio.h>
#include <vector>
#include <string>
#include <time.h>
#include <math.h>
#include <random> // The header for the generators.
#include <iomanip>
//Definitions=======================================================================
// Define the precision of real numbers, could be float/double.
#define real double
#define Pi 3.1415926535897932384626433832795
#define Zero 0
//typedef double2 Complex;
using namespace std;

struct Particle{
	real* xGPu;//save x position in GPU
	real* yGpu;//save y position in GPU
	int* cellX;//save xth cell of nth particle
	int* cellY;//save yth cell of nth particle
	int* cellList;//cell particle id for all particle, as [maxParticlePerCell*id + offsetCl]
	int* cellOffsetsCl;//offset of every cell list to save particle number in this cell 
	int* particleAroundId;//save ids around this on particle, use rd to judge wether is "around"
	int* particleAroundFlagX;//mask whether cell of idth particle at the edge of box
	int* particleAroundFlayY;//mask whether cell of idth particle at the edge of box
	int* offsetNL;//offset of every particle's neighbor list to save neighbor particle id
	int* NeighborList;//neighbor list
	int* NeighborListFlagX;//translate from particleAroundFlagX
	int* NeighborListFlagY;//translate from particleAroundFlagY
	real* fx;//force on the x direction
	real* fy;//force on the y direction
	real* x0ToUpdateHybridList;//save xGpu[id] to judge whether update hybrid list 
	real* y0ToUpdateHybridList;//save yGpu[id] to judge whether update hybrid list
} PT,pt;

struct Parameter{
	real boxX;//box size X
	real boxY;//box size Y
	real cellSizeX;//cell size in the x direction
	real cellSizeY;//cell size in the y direction
	int cellNumX;//num of cell in the x direction
	int cellNumY;//num of cell in the y direction
	real rho;		//�ܶ�
	int maxParticlePerCell;//theory maxmum particle number in one cell
	real rd;//deadline distance to get particleAroundId
	int mask0;//use for bit calculate
	int mask1;//use for bit calculate 	
	real miniInstanceBetweenParticle;//theory minimum distance from two particle
	real r0;//balance position
	real epsilon;//coefficient of force
	int kBT;//kB*T
	int gammaValue;//Viscosity coefficien
	real rOutUpdateList;//update hybrid list when any one particle move a distance greater than rOutUpdateList
	int paraticleNum; //������Ŀ
    real tStart;
    real tStop;
    real tStep;
} PM;
//Flag use to update hybrid list=================================================================================================
__global__ real updateHybridListFlag=0;

//input data=====================================================================================================
// ������������б�ṹ��
typedef struct NameList {
    const char* vName;    //������
    //char *vName;    //������
    void* vPtr;     //����ָ��
    VType vType;    //��������
    int vLen, vStatus; //�������ȡ�״̬
}NameList;

//�������
real boxX, boxY, cellSizeX, cellSizeY, rho, miniInstanceBetweenParticle, r0, epsilon;
int cellNumX, cellNumY, maxParticlePerCell, mask0, mask1, kBT, gammaValue, particleNum;
//�궨��
#define NameI(x) {#x,&x,N_I,sizeof(x)/sizeof(int)} //�������ͱ�������ָ�롢���͡����ȡ�״̬
#define NameR(x) {#x,&x,N_R,sizeof(x)/sizeof(real)}//����ʵ�ͱ�������ָ�롢���͡����ȡ�״̬

// ���������б�
NameList nameList[] = {
    NameR(boxX),
    NameR(boxY),
    NameR(cellSizeX),
    NameR(cellSizeY),
    NameR(rho),
    NameR(miniInstanceBetweenParticle),
    NameR(r0),
    NameR(epsilon),
    NameI(cellNumX),
    NameI(cellNumY),
    NameI(maxParticlePerCell),
    NameI(mask0),
    NameI(mask1),
    NameI(kBT),
    NameI(gammaValue),
    NameI(particleNum),

};
// ����꣬���ڼ򻯴���
#define NP_I ((int*)(nameList[k].vPtr) + j)
#define NP_R ((real*)(nameList[k].vPtr) + j)

//=======================================================================================================
__device__ void getCellList(int id, Particle PT, Parameter PM);
__device__ void getAroundCellParticleId(int id, Particle PT, Parameter PM);
__device__ void getNeighborList(int id, Particle PT, Parameter PM);
__device__ void updateHybridList(int id, Particle PT, Parameter PM);
__device__ void getForce(int id, Particle PT, Parameter PM);
__device__ void updatePosition(int id, Particle PT, Particle PM);
__device__ void checkAndUpdateHybridList(int id, Particle PT, Parameter PM);
__device__ void clearUpdateHybridListFlag(int id);
__global__ void iterate(Particle PT, Parameter PM);
__global__ int checkUpdateHybridList(int id, real xLast, real yLast, real xNow, real yNow, real rOutUpdateList);
void showProgress(real tNow, real tStart, real tStop, clock_t clockNow, clock_t clockStart);
int GetNameList(int argc, char** argv);
void PrintNameList(FILE* fp);
void Init_Coords(int flag, Particle PT, Parameter PM);
void Init_Parameter();
void Init_System(int argc, char** argv);
void MemAlloc();
void MemFree();
void HostUpdataToDevice();
void DeviceUpdataToHost();
void ExpoConf(const std::string& str_t);
//=================================================================================

int main(int argc, char** argv) {
    clock_t clockNow = clock();
    float tNow = PM.tStart;
    Init_System(argc, argv); //��ʼ��ϵͳ    
    HostUpdataToDevice();// �ϴ����ݵ��豸
    //gpu kernel
    for (tNow = PM.tStart;tNow < PM.tStop;tNow+=tStep) {
        iterate<<<1,PM.particleNum>>>(PT, PM,tNow-tStart);
        if (floor(tNow / P.tExpo) > floor((tNow - tStep) / P.tExpo)) {
            DeviceUpdataToHost();//�������ݵ�����
            int te = floor(tNow / P.tExpo);
            str_t = to_string(te);
            ExpoConf(str_t);
            showProgress(tNow,PM.tStart,PM.tStop,clockNow,clock());
    }
    MemFree();//�ͷ��ڴ�
    return 0;
}

//==========================================================================================================
__device__ void getCellList(int id,Particle PT,Parameter PM){
	int cellX[id]=std::floor(PT.xGpu[id]/PM.cellSizeX);
	int cellY[id]=std::floor(PT.yGpu[id]/PM.cellSizeY);
	int cellId=cellY*PM.cellNumX+cellX;
	int offsetCL=atomicAdd(&PT.cellOffsetsCL[cellId],1);
	if (offsetCL<PM.maxParticlePerCell){
		PT.cellList[cellId*PM.maxParticlePerCell+offsetCL]=id;
	}else{
		printf("wrong");//append cout error later
	}
	__syncthreads();
}

//==========================================================================================================
__device__ void getAroundCellParticleId(int id,Particle PT,Parameter PM){
	int offsetPAI=0;//particleAroundId put particleId in PAI
	int periodicBoundaryFlagX,periodicBoundaryFlagY;
	int cellXAround,cellYAround;
	int cellAroundId;
	for(int x=-1;x<=1;x++){
		for(int y=-1;y<=1;y++){
			//int cellXAround=cellX+x==-1?cellNumX-1:cellX+x==cellNumX?0:cellX+x;//periodic boundary condition
			//int cellYAround=cellY+y==-1?cellNumY-1:cellY+y==cellNumY?0:cellY+y;
			if(PT.cellX[id]+x==-1){
				cellXAround=cellNumX-1;
				periodicBoundaryFlagX=1;
			}else if(PT.cellX[id]+x==cellNumX){
				cellXAround=0;
				periodicBoundaryFlagX=-1;
					
			}else{
				cellXAround=PT.cellX[id]+x;
				periodicBoundaryFlagX=0;
			}
			if(PT.cellY[id]+y==-1){
				cellYAround=cellNumY-1;
				periodicBoundaryFlagY=1;
			}else if(PT.cellY[id]==cellNumY){
				cellYAround=0;
				periodicBoundaryFlagY=-1;
			}else{
				cellYAround=PT.cellY[id]+y;
				periodicBoundaryFlagY=0;
			}

			cellAroundId=cellYAround*PM.cellNumX+cellXAround;
			for(int i=0;i<cellOffsetsCL[cellAroundId]){
				PT.particleAroundId[offsetPAI]=PT.cellList[cellAroundId*PM.maxParticlePerCell+i];
				PT.particleAroundFlagX[offsetPAI]=periodicBoundaryFlagX;
				PT.particleAroundFlagY[offsetPAI]=periodicBoundaryFlagY;
				offsetPAI++;
			}
		}
	}
	__syncthreads();
}

//==========================================================================================================
__device__ void getNeighborList(int id,Particle PT,Parameter PM){
//get neighborList use bit calculate
	int AX=std::floor(xGpu[id]/PM.miniInstanceBetweenParticle);
	int AY=std::floor(yGpu[id]/PM.miniInstanceBetweenParticle);
	int A0=AY+(AX<<11);//make position real to int
	int A1=A0|PM.mask0;
	int iId,BX,BY,B0,B1,B2,A2,B2;
	int offsetNL;
	for(int i=0;i<offsetPAI;i++){
		iId=PT.particleAroundId[i];
		BX=std::floor(PT.xGpu[iId]/PM.miniIntstanceBetweenParticle);
		BY=std::floor(PT.yGpu[iId]/PM.miniInstanceBetweenParticle);
		B0=BY+(AY<<11);
		B1=B0|PM.mask0;
		A2=(A1-B0)&PM.mask1;
		B2=(B1-A0)&PM.mask1;
		if(!(( (A2&B2==0) | (A2&(B2<<1)) | (A2<<1&B2) )= 0)){
			offsetNL=atomicAdd(&PT.OffsetsNL[id],1);
			PT.NeighborList[id*PM.maxParticlePerNeighbor+offsetNL]=iId;
			PT.NeighborListFlagX[id*PM.maxParticlePerNeighbor+offsetNL]=PT.particleAroundFlagX[i];
			PT.NeighborListFlagY[id*PM.maxParticlePerNeighbor+offsetNL]=PT.particleAroundFlagY[i];
		}
	}
	__syncthreads();
}

//==========================================================================================================
__device__ void updateHybridList(int id,Particle PT,Parameter PM){
	PT.x0ToUpdateHybridList[id]=PT.xGpu[id];
	PT.y0ToUpdateHybridList[id]=PT.yGpu[id];
	getCellList(id,PT,PM);
	getAroundCellParticleId(id,PT,PM); 
	getNeighborList(id,PT,PM);
	__syncthreads();
}

//==========================================================================================================
__device__ void getForce(int id,Particle PT,Parameter PM){
	//get force
	real x,y,xi,yi,dx,dy,dr,f12;
	for(int i=0;i<PT.offsetNL[id];i++){
		x=PT.xGpu[id];
		y=PT.yGpu[id];
		xi=PT.xGpu[PT.NeighborList[id*PM.maxParticlePerNeighbor+i]];
		yi=PT.yGpu[PT.NeighborList[id*PM.maxParticlePerNeighbor+i]];
		dx=(x-xi+PT.NeighborListFlagX[i]*PM.boxX);
		dy=(y-yi+PT.NeighborListFlagY[i]*PM.boxY);
		dr=sqrt(dx*dx+dy*dy);
		f12=24*epsilon*pow(r0,6)*(2*pow(r0,6)-pow(dr,6))/pow(dr,14);
		PT.fx[id]+=f12*dx;
		PT.fy[id]+=f12*dy;
	}
	__syncthreads();
}

//==========================================================================================================
__device__ void updatePosition(int id,Particle PT,Particle PM){
	real fT=sqrt(2*kBT*gamma*tStep);
	xGpu[i]+=fmod((PT.fx[id]*tStep+fT*FRx)/PM.gamma+PM.boxX,PM.boxX);
	yGpu[i]+=fmod((PT.fy[id]*tStep+fT*FRy)/PM.gamma+PM.boxY,PM.boxY);
	__syncthreads();
}

//==========================================================================================================
__device__ void checkAndUpdateHybridList(int id,Particle PT,Parameter PM){
	checkUpdateHybridList(id,PT.x0ToUpdateHybridList[i],PT.y0ToUpdateHybridList[i],xGpu[i],yGpu[i]);
	__syncthreads();
	if(updateHybridListFlag){
		updateHybridList(id,PT,PM);
	}
	__syncthreads();
	clearUpdateHybridList(id);
	__syncthreads();//maybe wrong
}
	
//==========================================================================================================
__device__ void clearUpdateHybridListFlag(int id){
	if(id==0) updateHybridListFlag=0;
}

//==========================================================================================================
__global__ void iterate(Particle PT,Parameter PM,int startFlag){
    if (startFlag)updateHybridList(id, PT, PM);
	int id=blockIdx.y*blockDim.y+blockIdx.x//use one dimentional block ,every particle use one block
	checkAndUpdateHybridList(id,PT,PM);
	getForce(id,PT,PM);
	updatePostion(id,PT,PM);
}

//==========================================================================================================
__global__ int checkUpdateHybridList(int id,real xLast,real yLast,real xNow,real yNow,real rOutUpdateList){
	real dr2=(xNow-xLast)*(xNow-xLast)+(yNow-yLast)*(yNow-yLast);
	if(dr2<rOutUpdateList*rOutUpdateList){
		atomicExch(&dateHybridListFlag,1);
		return 1;	
	}else{
		return 0;
	}
}	

//==========================================================================================================
void showProgress(real tNow,real tStart,real tStop,clock_t clockNow,clock_t clockStart){
	std::cout.flush();
	real progress=((tNow-tStart)/(tStop-tStart);
	real tUsed=double(clockNow,clockStart)/CLOCKS_PER_SEC;
	real tUsePerdiction=tUsed/progress;
	printf("%f\%",progress*100);
	printf("   Peridict:%f",tUsePerdiction);
}

//==========================================================================================================
// ��ȡ�����б�ĺ���
int GetNameList(int argc, char** argv)
{
    int id, j, k, match, ok;
    char buff[80], * token;
    FILE* fp;

    strcpy(buff, argv[0]);
    strcat(buff, ".in");
    if ((fp = fopen(buff, "r")) == 0)return 0;
    for (k = 0; k < sizeof(nameList) / sizeof(NameList);k++)
        nameList[k].vStatus = 0;
    ok = 1;
    while (1) {
        fgets(buff, 80, fp);
        if (feof(fp))break;
        token = strtok(buff, " \t\n");
        if (!token)break;
        match = 0;
        for (k = 0; k < sizeof(nameList) / sizeof(NameList);k++) {
            if (strcmp(token, nameList[k].vName) == 0) {
                match = 1;
                if (nameList[k].vStatus == 0) {
                    nameList[k].vStatus = 1;
                    for (j = 0;j < nameList[k].vLen; j++) {
                        token = strtok(NULL, " \t\n");
                        if (token) {
                            switch (nameList[k].vType) {
                            case N_I:
                                *NP_I = atol(token);
                                break;
                            case N_R:
                                *NP_R = atof(token);
                                break;
                            }
                        }
                        else {   //���屨����Ϣ

                            nameList[k].vStatus = 2;
                            ok = 0;
                        }
                    }
                    token = strtok(NULL, ", \t\n");
                    if (token) {
                        nameList[k].vStatus = 3;
                        ok = 0;
                    }
                    break;
                }
                else {
                    nameList[k].vStatus = 4;
                    ok = 0;
                }
            }
        }
        if (!match)ok = 0;

    }
    fclose(fp);
    for (k = 0;k < sizeof(nameList) / sizeof(NameList);k++) {
        if (nameList[k].vStatus != 1)ok = 0;
    }
    return ok;
}

//========================================================================================
// ��ӡ�����б�ĺ���
void PrintNameList(FILE* fp)
{
    int j, k;

    fprintf(fp, "NameList --data\n");
    for (k = 0; k < sizeof(nameList) / sizeof(NameList);k++) {
        fprintf(fp, "%s\t", nameList[k].vName);
        if (strlen(nameList[k].vName) < 8)fprintf(fp, "\t");
        if (nameList[k].vStatus > 0) {
            for (j = 0;j < nameList[k].vLen;j++) {
                switch (nameList[k].vType) {
                case N_I:
                    fprintf(fp, "%d\t", *NP_I);
                    break;
                case N_R:
                    fprintf(fp, "%#g\t", *NP_R);
                    break;
                }
            }
        }
        switch (nameList[k].vStatus) {
        case 0:
            fprintf(fp, "--no data\n");
            break;
        case 1:
            break;
        case 2:
            fprintf(fp, "--missing data\n");
            break;
        case 3:
            fprintf(fp, "--extra data\n");
            break;
        case 4:
            fprintf(fp, "--multiple defined\n");
            break;
        }
        fprintf(fp, "\n");
    }
    fprintf(fp, "---------\n");
}

//========================================================================================
void Init_Coords(int flag, Particle PT, Parameter PM) {
    /*
    flag����ϵͳ�ĳ�ʼ����ʽ��flag=0������ȷֲ���flag=1��������ֲ�
    �����վ��ȷֲ�ʱ������������ܶȣ���ͬʱ���ճ�ʼ������Ŀ,��ʼϵͳ�����ںд�С��
    ����������ֲ�ʱ�������������Ŀ�����������������
    */
    //��ʼ���ںг���
    int N = PM.ParaticleNum;
    float rho = PM.rho;
    float L = sqrt(N / rho);
    //���������κ���
    float xBox = L;
    float yBox = L;
    PM.boxX = xBox;
    PM.boxY = yBox;
    int initUcell = sqrt(N); //��ʼx,y,����������Ŀ
    if flag == 0{
        float d_lattice = L / sqrt(N); //������
        //���ȷֲ� ϵͳ��ԭ��Ϊ����
        int n, nx, ny;
        n = 0;
        for (ny = 0;ny < initUcell; ny++) {
            for (nx = 0;nx < initUcell; nx++) {
                PT.xGPU[n] = nx * d_lattice;
                PT.yGPU[n] = ny * d_lattice;
                n++;
            }
        }
    }
    //����ֲ� ���ȷֲ��������������
    else if flag == 1{
        std::default_random_engine e;
        std::uniform_real_distribution<double> u(0.0, 1.0);
        e.seed(time(0));

        for (int n = 0; n < N; n++) {
            PT.xGPU[n] = u(e) * xBox;
            PT.yGPU[n] = u(e) * yBox;
        }
    }
}
//initial system=================================================================================================
void Init_Parameter() {
    PM.boxX = nameList.boxX;
    PM.boxY = nameList.boxY;
    PM.cellNumX = nameList.cellNumX;
    PM.cellNumX = nameList.cellNumX;
    PM.cellSizeX = nameList.cellSizeX;
    PM.cellSizeY = nameList.cellSizeY;
    PM.rho = nameList.rho;
    PM.maxParticlePerCell = nameList.maxParticlePerCell;
    PM.mask0 = nameList.mask0;
    PM.mask1 = nameList.mask1;
    PM.miniInstanceBetweenParticle = nameList.miniInstanceBetweenParticle;
    PM.r0 = nameList.r0;
    PM.epsilon = nameList.epsilon;
    PM.gammaValue = nameList.gammaValue;
    PM.kBT = nameList.kBT;
    PM.paraticleNum = nameList.paraticleNum;
    PM.mask0;
    PM.mask1;
}

void Init_System(int argc, char** argv) {
    GetNameList(argc, argv);//��������
    PrintNameList(stdout);//��ӡ��鵼������
    Init_Parameter();
    Init_Coords(0, PT, PM); //���ȷֲ�
    MemAlloc();//�����ڴ�
}

//mem ============================================================================================================

void MemAlloc() {
    // Allocate particle mem in host memory.
    pt.xGpu = new real[PM.paraticleNum];
    pt.yGpu = new real[PM.paraticleNum];
    pt.cellList = new int[PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell];
    pt.cellOffsetsCl = new int[PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell];
    pt.particleAroundId = new int[9 * PM.paraticleNum];
    pt.particleAroundFlagX = new int[PM.paraticleNum];
    pt.particleAroundFlayY = new int[PM.paraticleNum];
    pt.offsetNL = new int[PM.paraticleNum];
    pt.NeighborList = new int[PM.paraticleNum * PM.maxParticlePerCell];
    pt.NeighborListFlagX = new int[PM.paraticleNum];
    pt.NeighborListFlagY = new int[PM.paraticleNum];
    pt.fx = new real[PM.paraticleNum];
    pt.fy = new real[PM.paraticleNum];
    pt.x0ToUpdateHybridList = new int[PM.paraticleNum];
    pt.y0ToUpdateHybridList = new int[PM.paraticleNum];


    // Allocate memory of fields in device.
    hipMalloc((void**)&PT.cellX, PM.paraticleNum * sizeof(int));
    hipMalloc((void**)&PT.cellY, PM.paraticleNum * sizeof(int));
    hipMalloc((void**)&PT.cellList, PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell * sizeof(real));
    hipMalloc((void**)&PT.cellOffsetsCl, PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell * sizeof(real));
    hipMalloc((void**)&PT.particleAroundId, 9 * PM.paraticleNum * sizeof(int));
    hipMalloc((void**)&PT.particleAroundFlagX, PM.paraticleNum);
    hipMalloc((void**)&PT.particleAroundFlayY, PM.paraticleNum);
    hipMalloc((void**)&PT.offsetNL, PM.paraticleNum);
    hipMalloc((void**)&PT.NeighborList, PM.paraticleNum * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.NeighborListFlagX, PM.paraticleNum * sizeof(int));
    hipMalloc((void**)&PT.NeighborListFlagY, PM.paraticleNum * sizeof(int));
    hipMalloc((void**)&PT.fx, PM.paraticleNum * sizeof(real));
    hipMalloc((void**)&PT.fy, PM.paraticleNum * sizeof(real));
    hipMalloc((void**)&PT.x0ToUpdateHybridList, PM.paraticleNum * sizeof(real));
    hipMalloc((void**)&PT.y0ToUpdateHybridList, PM.paraticleNum * sizeof(real));
    hipMalloc((void**)&PT.xGpu, PM.paraticleNum * sizeof(real));
    hipMalloc((void**)&PT.yGpu, PM.paraticleNum * sizeof(real));
}
//===========================================================================
void MemFree() {
    // Free host memory
    delete[] pt.xGpu;
    delete[] pt.yGpu;
    delete[] pt.cellList;
    delete[] pt.cellOffsetsCl;
    delete[] pt.particleAroundId;
    delete[] pt.particleAroundFlagX;
    delete[] pt.particleAroundFlayY;
    delete[] pt.offsetNL;
    delete[] pt.NeighborList;
    delete[] pt.NeighborListFlagX;
    delete[] pt.NeighborListFlagY;
    delete[] pt.fx;
    delete[] pt.fy;
    delete[] pt.x0ToUpdateHybridList;
    delete[] pt.y0ToUpdateHybridList;

    // Free device memory
    hipFree(PT.xGpu);
    hipFree(PT.yGpu);
    hipFree(PT.cellX);
    hipFree(PT.cellY);
    hipFree(PT.cellList);
    hipFree(PT.cellOffsetsCl);
    hipFree(PT.particleAroundId);
    hipFree(PT.particleAroundFlagX);
    hipFree(PT.particleAroundFlayY);
    hipFree(PT.offsetNL);
    hipFree(PT.NeighborList);
    hipFree(PT.NeighborListFlagX);
    hipFree(PT.NeighborListFlagY);
    hipFree(PT.fx);
    hipFree(PT.fy);
    hipFree(PT.x0ToUpdateHybridList);
    hipFree(PT.y0ToUpdateHybridList);
}
//�ϴ�=============================================================================================
void HostUpdataToDevice() {
    hipMemcpy(pt.xGpu, PT.xGPU, PM.paraticleNum * sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(pt.xGpu, PT.xGPU, PM.paraticleNum * sizeof(real), hipMemcpyHostToDevice);
}
//����=============================================================================================
void DeviceUpdataToHost() {
    hipMemcpy(PT.xGPU, pt.xGpu, PM.paraticleNum * sizeof(real), hipMemcpyDeviceToHost);
    hipMemcpy(PT.yGPU, pt.yGpu, PM.paraticleNum * sizeof(real), hipMemcpyDeviceToHost);
}
//output===========================================================================================

void ExpoConf(const std::string& str_t) {
    std::ofstream ConfFile;
    //�����������
    int PrecData = 8;

    // �ļ���
    std::string ConfFileName = "data/conf_" + str_t + ".dat";
    ConfFile.open(ConfFileName.c_str());

    if (!ConfFile.is_open()) {
        std::cerr << "�޷����ļ�: " << ConfFileName << std::endl;
        return;
    }
    for (int idx = 0; idx < PM.paraticleNum; idx++) {
        // ʹ�ù̶���ʽ�;����������
        ConfFile << std::fixed << std::setprecision(PrecData)
            << PT.xGpu[idx] << ' '
            << PT.yGpu[idx];
        ConfFile << std::endl; // ����
    }

    ConfFile.close();
}


